#include "hip/hip_runtime.h"
#include "kernel.h"

#include <cstdio>

void __global__ print()
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    std::printf("%d\n", idx);
}

void f()
{
    print<<<1, 10>>>();
    hipDeviceSynchronize();
}