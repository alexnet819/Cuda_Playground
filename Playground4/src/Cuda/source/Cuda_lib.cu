#include "hip/hip_runtime.h"
#include "Cuda_lib.cuh"
//#include "../include/Cuda_lib.cuh"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include<thrust/host_vector.h>
#include<thrust/device_vector.h>
#include <hip/hip_runtime.h>
#include <cstdio>

#define MAX 100


//__global__ void kernel_Random(unsigned int seed, int *result) {
__global__ void kernel_Random() {
	// hiprandState_t states;
    // int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // hiprand_init(seed, 0, 0, &states);
    // *result = hiprand_normal(&states);
	//hiprandState_t state;

    //hiprand_init(seed, 0, 0, &state);
    //std::printf("\n%f", hiprand_normal(&state));

}

cuda_lib::cuda_lib() {
}

cuda_lib::~cuda_lib() {
}

void cuda_lib::random(unsigned int seed, int* result) {
	std::cout << "test" << std::endl;
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("CUDA error: %s\n", hipGetErrorString(err));
		return;
	}
}

// void cuda_lib::random(unsigned int seed, int* result) {
// 	std::cout << "test" << std::endl;
// 	hipError_t err = hipGetLastError();
// 	if (err != hipSuccess) {
// 		printf("CUDA error: %s\n", hipGetErrorString(err));
// 		return;
// 	}
	
// 	int *gpu1 = 0;
// 	unsigned int gpu_seed = 0;
// 	hipMalloc((void**)&gpu1, sizeof(int));
// 	err = hipGetLastError();
// 	if (err != hipSuccess) {
// 		printf("CUDA error: %s\n", hipGetErrorString(err));
// 		return;
// 	}
// 	hipMalloc((void**)&gpu_seed, sizeof(unsigned int));
// 	err = hipGetLastError();
// 	if (err != hipSuccess) {
// 		printf("CUDA error: %s\n", hipGetErrorString(err));
// 		return;
// 	}
// 	std::cout << "test" << std::endl;
// 	hipMemcpy(gpu1, result, sizeof(int), hipMemcpyHostToDevice);
// 	err = hipGetLastError();
// 	if (err != hipSuccess) {
// 		printf("CUDA error: %s\n", hipGetErrorString(err));
// 		return;
// 	}

// 	hipMemcpy(&gpu_seed, &seed, sizeof(unsigned int), hipMemcpyHostToDevice);
// 	err = hipGetLastError();
// 	if (err != hipSuccess) {
// 		printf("CUDA error: %s\n", hipGetErrorString(err));
// 		return;
// 	}
// 	std::cout << "test" << std::endl;
// 	//kernel_Random<<<1,1>>>(gpu_seed, gpu1);
// 	dim3 blockSize(1);
// 	dim3 numBlocks(1);
// 	//kernel_Random<<<numBlocks, blockSize>>>();
// 	//hipError_t err = hipGetLastError();
// 	// hiprandStatus_t _t;
// 	// hiprandCreateGenerator(&_t, HIPRAND_RNG_PSEUDO_DEFAULT);
// 	hiprandGenerator_t gen;
// 	hiprandStatus_t status = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
// 	hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
// 	hiprandGenerateNormalDouble(gen, (double *)gpu1, 1, 100, 1.0);
// 	// err = hipGetLastError();
// 	// if (err != hipSuccess) {
// 	// 	printf("CUDA error: %s\n", hipGetErrorString(err));
// 	// 	return;
// 	// }
// 	hipDeviceSynchronize();
// 	std::cout << "test" << std::endl;
// 	hipMemcpy(result, gpu1, sizeof(int), hipMemcpyDeviceToHost);
// 	std::cout << result << std::endl;
// 	//hipMemcpy(gpu1, result, sizeof(int), hipMemcpyDeviceToHost);
// 	hipFree(gpu1);
// 	hipFree(&gpu_seed);
// 	hipDeviceSynchronize();
// 	hiprandDestroyGenerator(gen);
// }

void cuda_lib::vectorAdd(int *a, int *b, int n) {
}