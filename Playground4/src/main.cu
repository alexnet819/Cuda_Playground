#include "hip/hip_runtime.h"
/* random2.cu */

#include <unistd.h>
#include <cstdio>
#include <typeinfo>

/* we need these includes for CUDA's random number stuff */
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include<thrust/host_vector.h>
#include<thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/random.h>

// #include "Cuda/include/Cuda_lib.cuh"
#include "Cuda_lib.cuh"
// #include "include/Cuda_lib.cuh"


// /* this GPU kernel function calculates a random number and stores it in the parameter */
// __global__ void random(unsigned int seed, int* result) {
//     /* CUDA's random number library uses hiprandState_t to keep track of the seed value
//        we will store a random state for every thread  */
//     hiprandState_t state;

//     /* we have to initialize the state */
//     hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
//             0, /* the sequence number is only important with multiple cores */
//             0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
//             &state);

//     /* hiprand works like rand - except that it takes a state as a parameter */
//     *result = hiprand(&state) % MAX;
// }

template<typename T>
struct RandGen
{
    RandGen(T min, T max):_min(min), _max(max) {}
	T _min;
	T _max;
    __device__
    double operator () (int idx)
    {
        thrust::default_random_engine randEng(1337);
        thrust::uniform_real_distribution<double> uniDist(_min, _max);
		randEng.discard(idx);
        return uniDist(randEng);
    }
};

int main() {

	// cuda_lib instance;
	// /* host copy of result */
	// thrust::host_vector<double> h_result(10 ,0);
	// thrust::device_vector<double> d_result(10, 0);
	// thrust::default_random_engine rng(1337);
	// thrust::uniform_real_distribution<double> dist(-50.0, 50.0);
	// thrust::generate(d_result.begin(), d_result.end(), [&] { return dist(rng); });

	// std::cout << "test" << std::endl;

	// //instance.random(0, d_result);

	// //d_result.copy(h_result.begin(), h_result.end());
	// /* print the result */
	// for(int i = 0;i<h_result.size();i++){
	// 	std::cout << "Random number: " << h_result[i] << std::endl;

	// }

	const int num = 1000;
	thrust::device_vector<double> rVec(num);
	std::cout << (int)rVec.size() << std::endl;
	std::cout << typeid(rVec.size()).name() << std::endl;
	thrust::transform(thrust::make_counting_iterator(0), thrust::make_counting_iterator((int)rVec.size()), rVec.begin(), RandGen<double>(0, 1));

	for(int i = 0;i<rVec.size();i++)
		std::cout << "Random number: " << rVec[i] << std::endl;
    return 0;
}
