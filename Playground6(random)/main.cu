#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

__global__ void init(unsigned int seed, hiprandState_t* states) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}

__global__ void randoms(hiprandState_t* states, int* numbers) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    numbers[idx] = hiprand(&states[idx]) % 100;
}

int main() {
    int N = 100;
    hiprandState_t* states;
    int* numbers;
    hipMalloc((void**)&states, N * sizeof(hiprandState_t));
    hipMalloc((void**)&numbers, N * sizeof(int));

    init<<<N, 1>>>(time(0), states);
    randoms<<<N, 1>>>(states, numbers);


	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		printf("CUDA error: %s\n", hipGetErrorString(err));
		return;
	}
	hipDeviceSynchronize();
    int h_numbers[N];
    hipMemcpy(h_numbers, numbers, N * sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++) {
        printf("%d ", h_numbers[i]);
    }

    hipFree(states);
    hipFree(numbers);

    return 0;
}