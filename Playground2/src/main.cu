#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<thrust/host_vector.h>
#include<thrust/device_vector.h>
#include<vector>
#include<iostream>

// __global__はGPUで実行される関数であることを示す
__global__ void vectorAdd(int *a, int *b, int n) {
	int thread = blockIdx.x * blockDim.x + threadIdx.x;
	for(int j = 0; j < n; j++)
		a[thread] = thread * b[j];
}
int main(int argc, char** argv) {

	// GPUにデータを渡すようにvectorを作成
	// NVIDIAがstd::vector相当のライブラリを用意してくれている
	int *vec1[10], *vec2[20];
	//thrust::host_vector<int> vec1(10);
	//thrust::host_vector<int> vec2(20);
	
	// vectorに値を代入する
	for (int i = 0; i < 20; i++) {
		if(i < 10)
			vec1[i] = i;
		vec2[i] = i * 2;
	}

	// vec1の中身を表示
	std::cout << "Vector 1: ";
	for (int i = 0; i < vec1.size(); i++)
	{
		std::cout << vec1[i] << " ";
	}
	
	std::cout << std::endl;

	// GPUにvec1, vec2のデータを渡す
	int *gpu_vec1, *gpu_vec2;
	hipMalloc((void**)&gpu_vec1, sizeof(vec1) * sizeof(int));
	hipMalloc((void**)&gpu_vec2, sizeof(vec2) * sizeof(int));
	hipMemcpy(gpu_vec1, vec1, sizeof(vec1) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(gpu_vec2, vec2, sizeof(vec2) * sizeof(int), hipMemcpyHostToDevice);


	//thrust::device_vector<int> gpu_vec1 = vec1;
	//thrust::device_vector<int> gpu_vec2 = vec2;

	// GPUで起動するthreadの数を決める
	dim3 block(sizeof(vec1) / sizeof(int));
	dim3 grid(1);

	// GPUでvectorAdd関数を実行
	vectorAdd <<<grid, block>>>(gpu_vec1, gpu_vec2, sizeof(vec2) / sizeof(int);

	// GPUで計算した結果をvec1にコピー
	hipMemcpy(vec1, gpu_vec1, sizeof(vec1) * sizeof(int), hipMemcpyDeviceToHost);
	std::cout << "Vector 1: ";
	for (int i = 0; i < vec1.size(); i++)
	{
		std::cout << vec1[i] << " ";
	}

	std::cout << std::endl;
	return 0;
}