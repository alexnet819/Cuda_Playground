#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/random.h>

#include <random>
#include <ctime>
#include <typeinfo>


template<typename U>
struct RandGen;
	// int型のための特殊化
template<>
struct RandGen<int>
{
	int _min, _max, _time;
	RandGen(int min, int max, std::uint_fast64_t time):_min(min), _max(max), _time(time) {}
	__host__ __device__
	int operator () (int idx) {
		thrust::default_random_engine randEng(_time);
		thrust::uniform_int_distribution<int> uniDist(_min, _max);
		randEng.discard(idx);
		return uniDist(randEng);
	}
};
// double型のための特殊化
template<>
struct RandGen<double>
{
	double _min, _max;
	int _time;
	RandGen(double min, double max, std::uint_fast64_t time):_min(min), _max(max), _time(time) {}
	__host__ __device__
	double operator () (int idx)
	{
		thrust::default_random_engine randEng(_time);
		thrust::uniform_real_distribution<double> uniDist(_min, _max);
		randEng.discard(idx);
		return uniDist(randEng);
	}
};

// float型のための特殊化
template<>
struct RandGen<float>
{
	float _min, _max;
	int _time;
	RandGen(float min, float max, int time):_min(min), _max(max), _time(time) {}
	__host__ __device__
	float operator () (int idx)
	{
		thrust::default_random_engine randEng(_time);
		thrust::uniform_real_distribution<float> uniDist(_min, _max);
		randEng.discard(idx);
		return uniDist(randEng);
	}
};

template<typename T>
class CudaRandom {
public:
	CudaRandom() {}
	~CudaRandom() {}

	void HostRand(thrust::host_vector<T>& host_vec, T min = 0, T max = 1) {
		std::random_device rd;
		_mt.seed(rd());
		thrust::transform(thrust::make_counting_iterator(0), thrust::make_counting_iterator((int)host_vec.size()), host_vec.begin(), RandGen<T>(min, max, _mt()));
		// thrust::default_random_engine rng(1337);
		// thrust::uniform_real_distribution<double> dist(min, max);
		// thrust::generate(host_vec.begin(), host_vec.end(), [&] { return dist(rng); });
		// return host_vec;
	}

	void GPURand(thrust::device_vector<T>& device_vec, T min = 0, T max = 1) {
		std::random_device rd;
		_mt.seed(rd());
		thrust::transform(thrust::make_counting_iterator(0), thrust::make_counting_iterator((int)device_vec.size()), device_vec.begin(), RandGen<T>(min, max, _mt()));
		//return device_vec;
	}
private:
	std::mt19937_64 _mt;
};

