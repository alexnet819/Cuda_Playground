#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include "Cuda_Random.cu"

// __global__ void init(unsigned int seed, hiprandState_t* states) {
//     int idx = threadIdx.x + blockIdx.x * blockDim.x;
//     hiprand_init(seed, idx, 0, &states[idx]);
// }

// __global__ void randoms(hiprandState_t* states, int* numbers) {
//     int idx = threadIdx.x + blockIdx.x * blockDim.x;
//     numbers[idx] = hiprand(&states[idx]) % 100;
// }

// int main() {
//     int N = 100;
//     hiprandState_t* states;
//     int* numbers;
//     hipMalloc((void**)&states, N * sizeof(hiprandState_t));
//     hipMalloc((void**)&numbers, N * sizeof(int));

//     init<<<N, 1>>>(time(0), states);
//     randoms<<<N, 1>>>(states, numbers);


// 	hipError_t err = hipGetLastError();
// 	if (err != hipSuccess) {
// 		printf("CUDA error: %s\n", hipGetErrorString(err));
// 		return;
// 	}
// 	hipDeviceSynchronize();
//     int h_numbers[N];
//     hipMemcpy(h_numbers, numbers, N * sizeof(int), hipMemcpyDeviceToHost);

//     for(int i = 0; i < N; i++) {
//         printf("%d ", h_numbers[i]);
//     }

//     hipFree(states);
//     hipFree(numbers);

//     return 0;
// }

int main() {
	using rand_type = double;
	thrust::host_vector<rand_type> host_vec(10, 0);
	thrust::device_vector<rand_type> device_vec(10, 0);
	CudaRandom<rand_type> cr;

	// for (int i = 0; i < 10; i++){
	// 	printf("%f \n", static_cast<double>(device_vec[i]));
	// }
	//thrust::fill(device_vec.begin(), device_vec.end(), 0);
	// host_vec = cr.HostRand(host_vec, 0, 1);
	// device_vec = cr.GPURand(device_vec, 0, 1);
	//cr.HostRand(host_vec, 0, 10);
	cr.GPURand(device_vec, 0, 100);

	// for (int i = 0; i < 10; i++){
	// 	std::cout << host_vec[i] << std::endl;
	// 	//printf("%f ", static_cast<double>(host_vec[i]));
	// 	//printf("\n");
	// }
	for (int i = 0; i < device_vec.size(); i++){
		std::cout << device_vec[i] << std::endl;
		//printf("%d \n", static_cast<int>(device_vec[i]));
	}
    return 0;
}